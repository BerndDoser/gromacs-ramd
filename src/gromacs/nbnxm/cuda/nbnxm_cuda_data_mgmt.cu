/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2012,2013,2014,2015,2016 by the GROMACS development team.
 * Copyright (c) 2017,2018,2019,2020,2021, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \file
 *  \brief Define CUDA implementation of nbnxn_gpu_data_mgmt.h
 *
 *  \author Szilard Pall <pall.szilard@gmail.com>
 */
#include "gmxpre.h"

#include <assert.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>

// TODO We would like to move this down, but the way NbnxmGpu
//      is currently declared means this has to be before gpu_types.h
#include "nbnxm_cuda_types.h"

// TODO Remove this comment when the above order issue is resolved
#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/device_context.h"
#include "gromacs/gpu_utils/gpu_utils.h"
#include "gromacs/gpu_utils/gpueventsynchronizer.cuh"
#include "gromacs/gpu_utils/pmalloc.h"
#include "gromacs/hardware/device_information.h"
#include "gromacs/hardware/device_management.h"
#include "gromacs/math/vectypes.h"
#include "gromacs/mdlib/force_flags.h"
#include "gromacs/mdtypes/interaction_const.h"
#include "gromacs/mdtypes/md_enums.h"
#include "gromacs/nbnxm/atomdata.h"
#include "gromacs/nbnxm/gpu_data_mgmt.h"
#include "gromacs/nbnxm/gridset.h"
#include "gromacs/nbnxm/nbnxm.h"
#include "gromacs/nbnxm/nbnxm_gpu.h"
#include "gromacs/nbnxm/nbnxm_gpu_data_mgmt.h"
#include "gromacs/nbnxm/pairlistsets.h"
#include "gromacs/pbcutil/ishift.h"
#include "gromacs/timing/gpu_timing.h"
#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/cstringutil.h"
#include "gromacs/utility/fatalerror.h"
#include "gromacs/utility/real.h"
#include "gromacs/utility/smalloc.h"

#include "nbnxm_cuda.h"

namespace Nbnxm
{

/* This is a heuristically determined parameter for the Kepler
 * and Maxwell architectures for the minimum size of ci lists by multiplying
 * this constant with the # of multiprocessors on the current device.
 * Since the maximum number of blocks per multiprocessor is 16, the ideal
 * count for small systems is 32 or 48 blocks per multiprocessor. Because
 * there is a bit of fluctuations in the generated block counts, we use
 * a target of 44 instead of the ideal value of 48.
 */
static unsigned int gpu_min_ci_balanced_factor = 44;

void gpu_init_platform_specific(NbnxmGpu* /* nb */)
{
    /* set the kernel type for the current GPU */
    /* pick L1 cache configuration */
    cuda_set_cacheconfig();
}

void gpu_free(NbnxmGpu* nb)
{
    if (nb == nullptr)
    {
        return;
    }

    delete nb->timers;
    sfree(nb->timings);

    NBAtomData* atdat   = nb->atdat;
    NBParamGpu* nbparam = nb->nbparam;

    if (nbparam->elecType == ElecType::EwaldTab || nbparam->elecType == ElecType::EwaldTabTwin)
    {
        destroyParamLookupTable(&nbparam->coulomb_tab, nbparam->coulomb_tab_texobj);
    }

    if (!useLjCombRule(nb->nbparam->vdwType))
    {
        destroyParamLookupTable(&nbparam->nbfp, nbparam->nbfp_texobj);
    }

    if (nbparam->vdwType == VdwType::EwaldGeom || nbparam->vdwType == VdwType::EwaldLB)
    {
        destroyParamLookupTable(&nbparam->nbfp_comb, nbparam->nbfp_comb_texobj);
    }

    freeDeviceBuffer(&atdat->shiftVec);
    freeDeviceBuffer(&atdat->fShift);

    freeDeviceBuffer(&atdat->eLJ);
    freeDeviceBuffer(&atdat->eElec);

    freeDeviceBuffer(&atdat->f);
    freeDeviceBuffer(&atdat->xq);
    if (useLjCombRule(nb->nbparam->vdwType))
    {
        freeDeviceBuffer(&atdat->ljComb);
    }
    else
    {
        freeDeviceBuffer(&atdat->atomTypes);
    }

    /* Free plist */
    auto* plist = nb->plist[InteractionLocality::Local];
    freeDeviceBuffer(&plist->sci);
    freeDeviceBuffer(&plist->cj4);
    freeDeviceBuffer(&plist->imask);
    freeDeviceBuffer(&plist->excl);
    delete plist;
    if (nb->bUseTwoStreams)
    {
        auto* plist_nl = nb->plist[InteractionLocality::NonLocal];
        freeDeviceBuffer(&plist_nl->sci);
        freeDeviceBuffer(&plist_nl->cj4);
        freeDeviceBuffer(&plist_nl->imask);
        freeDeviceBuffer(&plist_nl->excl);
        delete plist_nl;
    }

    /* Free nbst */
    pfree(nb->nbst.eLJ);
    nb->nbst.eLJ = nullptr;

    pfree(nb->nbst.eElec);
    nb->nbst.eElec = nullptr;

    pfree(nb->nbst.fShift);
    nb->nbst.fShift = nullptr;

    delete atdat;
    delete nbparam;
    delete nb;

    if (debug)
    {
        fprintf(debug, "Cleaned up CUDA data structures.\n");
    }
}

int gpu_min_ci_balanced(NbnxmGpu* nb)
{
    return nb != nullptr ? gpu_min_ci_balanced_factor * nb->deviceContext_->deviceInfo().prop.multiProcessorCount
                         : 0;
}

void* gpu_get_xq(NbnxmGpu* nb)
{
    assert(nb);

    return static_cast<void*>(nb->atdat->xq);
}

DeviceBuffer<gmx::RVec> gpu_get_f(NbnxmGpu* nb)
{
    assert(nb);

    return reinterpret_cast<DeviceBuffer<gmx::RVec>>(nb->atdat->f);
}

DeviceBuffer<gmx::RVec> gpu_get_fshift(NbnxmGpu* nb)
{
    assert(nb);

    return reinterpret_cast<DeviceBuffer<gmx::RVec>>(nb->atdat->fShift);
}

} // namespace Nbnxm
